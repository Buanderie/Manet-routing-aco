#include "hip/hip_runtime.h"
// #include <hiprand.h>
// #include <hiprand/hiprand_kernel.h>

#include <cstdlib>
#include <ctime>
#include <cmath>
#include <cstring>
#include <memory>
#include <iostream>
#include <vector>
#include <algorithm>
#include <iomanip>
#include <hiprand/hiprand_kernel.h>

#include "Packet.h"
#include "RoutingEntry.hpp"
#include "config.h"
#include "hipError_t.h"


class PacketBuffer
{
public:
    
    unsigned size_;
    unsigned num_of_elements_;
    Packet* buffer_;

    __device__
    PacketBuffer() : num_of_elements_(0), size_(0), buffer_(NULL) {}

    __device__
    ~PacketBuffer() 
    {
        if (buffer_ != NULL)
        {
            delete[] buffer_;
        }
    }

    __device__
    void addPacket(Packet& packet)
    {
        if (buffer_ == NULL)
        {
            size_ = BUFFER_STEP;
            buffer_ = new Packet[size_];
        }
        else if (num_of_elements_ == size_)
        {
            size_ += BUFFER_STEP;
            Packet* new_buffer = new Packet[size_];
            memcpy(new_buffer, buffer_, num_of_elements_);
            delete[] buffer_;
            buffer_ = new_buffer;
        }

        buffer_[num_of_elements_] = packet;
        num_of_elements_ += 1;
    }

    __device__
    void reset()
    {
        delete[] buffer_;
        buffer_ = NULL;
        size_ = 0;
        num_of_elements_ = 0;
    }
};

__device__ 
int calcElem(int node, int target, int neighbour)
{
    return (node*nodes_num + target)*nodes_num  + neighbour;
}

__device__
int calcBufferElem(int node, int neighbour)
{
    return node*nodes_num + neighbour;
}


__device__
int drawNextHop(RoutingEntry* routing_for_neighbours, int node, int target, hiprandState& state)
{
    float pheromoneSum = 0.0;

    //count sum of pheromones for available routes
    for (int i=0; i < nodes_num; ++i)
    {
        int elem = calcElem(node, target, i);
        float pheromone = routing_for_neighbours[elem].pheromone;
        if (pheromone > 0.0)
        {
            pheromoneSum += pheromone;
        }
    }

    float part = hiprand_uniform(&state);
    float random = part * pheromoneSum;

    int choosenHop = -1;
    while(random > 0.0) {
        ++choosenHop;
        int elem = calcElem(node, target, choosenHop);
        float pheromone = routing_for_neighbours[elem].pheromone;
        if (pheromone > 0.0) // if is my neighbour
        {
            random -=  pheromone;
        }
    }

    return choosenHop;
}



// TO się musi wykonać sekwencyjnie. Czasu nie oszukasz. Ale może obejdzie się bez kopiowania do hosta
// [node][target][neighbour]

__global__
void initializePacketBuffers(PacketBuffer* incomming_buffers, 
                             PacketBuffer* outgoing_buffers)
{
    const int node = blockIdx.x;
    for (int i=threadIdx.x; i < gridDim.x; i += blockDim.x)
    {
        new (incomming_buffers + calcBufferElem(node, i)) PacketBuffer();
        new (outgoing_buffers + calcBufferElem(i, node)) PacketBuffer();
    }
}

__global__
void deinitializePacketBuffers(PacketBuffer* incomming_buffers, 
                               PacketBuffer* outgoing_buffers)
{
    const int node = blockIdx.x;
    for (int i=threadIdx.x; i < gridDim.x; i += blockDim.x)
    {
        incomming_buffers[calcBufferElem(node, i)].~PacketBuffer();
        outgoing_buffers[calcBufferElem(i, node)].~PacketBuffer();
    }
}

__global__
void nodesTick(PacketBuffer* incomming_buffers, 
               PacketBuffer* outgoing_buffers, 
               RoutingEntry* routing_table,
               int from, int to, unsigned tick)
{
    const int node = blockIdx.x;
    int tId = threadIdx.x + (blockIdx.x * blockDim.x);
    hiprandState state;
    hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

    
    //TODO add packet generation randomly
    if (node == from && threadIdx.x == 0 && tick%5 == 0)
    {
        PacketBuffer& buffer = incomming_buffers[calcBufferElem(from, from)];
        Packet packet(from, to, tick);
        buffer.addPacket(packet);
    }
    
    if (node == to && threadIdx.x == 0 && tick%5 == 0)
    {
        PacketBuffer& buffer = incomming_buffers[calcBufferElem(to, to)];
        Packet packet(to, from, tick);
        buffer.addPacket(packet);
    }
        

    __syncthreads();


    // Ewaporacja tablicy feromonów
    int start_elem = node * nodes_num * nodes_num;
    int end_elem = start_elem + nodes_num * nodes_num;
    for (int elem = start_elem + threadIdx.x; elem < end_elem; elem += blockDim.x)
    {
        RoutingEntry& routing_entry = routing_table[elem];
        if (routing_entry.pheromone > 0.0)
        {
            routing_entry.evaporatePheromone();
        }
    }


    __syncthreads();
    

    // Updating next_hops TODO could take packets more evenly
    for (int prev_hop = 0; prev_hop < gridDim.x; ++prev_hop) //neighbours from which we reived packets
    {
        PacketBuffer& current_incoming_packet_buffer = incomming_buffers[calcBufferElem(node, prev_hop)];

        for (int i=threadIdx.x; i< current_incoming_packet_buffer.num_of_elements_; i += blockDim.x)
        {
            Packet& current_packet = current_incoming_packet_buffer.buffer_[i];

            if (prev_hop != node)
            {
                float pheromone_increasement = RoutingEntry::calculateIncreasement(current_packet.hops_count);
                RoutingEntry& routing_entry = routing_table[calcElem(node, current_packet.sourceAddress, prev_hop)];
                atomicAdd(&routing_entry.pheromone, pheromone_increasement);
            }

            if (node == current_packet.destinationAddress)
            {
                //Since current_packet.next_hop == node, this packet won't be copied any more.

                // std::cout << "Packet " << current_packet.sequenceNumber << " reached node " << node << " after " << current_packet.hops_count << " hops." << std::endl;
            }
            else
            {
                int next_hop = drawNextHop(routing_table, node, current_packet.destinationAddress, state);
                current_packet.hops_count += 1;
                current_packet.next_hop = next_hop;

                float pheromone_increasement = RoutingEntry::calculateIncreasement(current_packet.hops_count);
                RoutingEntry& routing_entry = routing_table[calcElem(node, current_packet.destinationAddress, next_hop)];
                atomicAdd(&routing_entry.pheromone, pheromone_increasement);

                // std::cout << "At node " << node << " packet " << current_packet.sequenceNumber << " whose target is " << current_packet.destinationAddress << " came from " << prev_hop << " and will be send to " << next_hop << "." << std::endl;

            }

        }
    }

   
    __syncthreads();
    

    // Czyszczenie buforów wyjściowych i wysyłanie pakietów
    for (int next_hop = threadIdx.x; next_hop < gridDim.x; next_hop += blockDim.x)
    {
        PacketBuffer& outgoing_packet_buffer = outgoing_buffers[calcBufferElem(next_hop, node)];
        outgoing_packet_buffer.reset();

        for (int prev_hop = 0; prev_hop < gridDim.x; ++prev_hop) //neighbours from which we reived packets
        {
            PacketBuffer& incoming_packet_buffer = incomming_buffers[calcBufferElem(node, prev_hop)];

            for (int i=0; i< incoming_packet_buffer.num_of_elements_; ++i)
            {
                Packet& current_packet = incoming_packet_buffer.buffer_[i];

                if (current_packet.next_hop == next_hop && current_packet.destinationAddress != node)
                {
                    outgoing_packet_buffer.addPacket(current_packet);
                    // std::cout << "At node " << node << " packet " << current_packet.sequenceNumber << " whose target is " << current_packet.destinationAddress << " came from " << prev_hop << " and goes to " << next_hop << "." << std::endl;
                }
            }
        }
    } 

    
    __syncthreads();
    

    //free incoming buffer
    for (int prev_hop = threadIdx.x; prev_hop < gridDim.x; prev_hop += blockDim.x)
    {
        PacketBuffer& current_incoming_packet_buffer = incomming_buffers[calcBufferElem(node, prev_hop)];
        current_incoming_packet_buffer.reset();
    }

}














int printBestPath(int from, int to, RoutingEntry (&routing_table)[nodes_num][nodes_num][nodes_num])
{
    int counter = 0;
    std::vector<int> visited;
    int current_hop = from;
    std::cout << current_hop << ", ";
    while (current_hop != to)
    {
        if (std::find(visited.begin(), visited.end(), current_hop) != visited.end())
        {
            std::cout << "Loop";
            break;
        }
        visited.push_back(current_hop);

        int best_hop = -1;
        float best_pheromone = -10000.0f;
        for (int i=0; i < nodes_num; ++i)
        {
            float pheromone = routing_table[current_hop][to][i].pheromone;
            if (pheromone > best_pheromone)
            {
                best_hop = i;
                best_pheromone = pheromone;
            }
        }

        current_hop = best_hop;
        counter += 1;
        std::cout << current_hop << ", ";
    }
    std::cout << " (Path length: " << counter << ")" << std::endl;
    return counter;
}

void printRoutingTable(RoutingEntry (&routing_table)[nodes_num][nodes_num][nodes_num])
{
    for (int node=0; node < nodes_num; ++node)
    {
        std::cout << "\n\nNODE: " << node;
        for (int target=0; target < nodes_num; ++target)
        {
            std::cout << "\n  target " << target << ": ";
            for (int neighbour=0; neighbour < nodes_num; ++neighbour)
            {
                // std::cout.width(9);
                std::cout << std::setw( 11 ) << routing_table[node][target][neighbour].pheromone << " ";
            }
        }
    }
    std::cout << std::endl;
}


void initializeRoutingTable(std::string file_name, RoutingEntry (&routing_table)[nodes_num][nodes_num][nodes_num]);

//liczba bloków to liczba węzłów. Węzłów może być aż do 2^16-1, czyli 65535
//liczba wątków to maksymalna liczba pakietów do przetworzenia na raz. Moja CUDA umożliwia stworzenie 1024 wątków.
int main()
{
    // PacketBuffer incomming_buffer[nodes_num][nodes_num];
    // PacketBuffer outgoing_buffer[nodes_num][nodes_num];
    RoutingEntry routing_table[nodes_num][nodes_num][nodes_num];

    // initialize values
    initializeRoutingTable("graph.json", routing_table);




    PacketBuffer* device_incomming_buffer_ptr;
    PacketBuffer* device_outgoing_buffer_ptr;
    RoutingEntry* device_routing_table_ptr;

    CudaSafeCall( hipMalloc(&device_incomming_buffer_ptr ,buffers_elem_num*sizeof(PacketBuffer)) );
    CudaSafeCall( hipMalloc(&device_outgoing_buffer_ptr, buffers_elem_num*sizeof(PacketBuffer)) );
    CudaSafeCall( hipMalloc(&device_routing_table_ptr, total_elem_num*sizeof(RoutingEntry)) );

    // hipMemcpy(device_incomming_buffer_ptr, incomming_buffer, buffers_elem_num*sizeof(PacketBuffer), hipMemcpyHostToDevice);
    // hipMemcpy(device_outgoing_buffer_ptr, outgoing_buffer, buffers_elem_num*sizeof(PacketBuffer), hipMemcpyHostToDevice);
    CudaSafeCall( hipMemcpy(device_routing_table_ptr, routing_table, total_elem_num*sizeof(RoutingEntry), hipMemcpyHostToDevice) );




    const unsigned from = 0;
    const unsigned to = 2;

    initializePacketBuffers<<<nodes_num, thread_num>>>(device_incomming_buffer_ptr, 
                                                       device_outgoing_buffer_ptr);
    CudaCheckError();

    for(int ticks=0; ticks<10000; ++ticks)
    {
        if (ticks%200 == 0)
        {
            CudaSafeCall( hipMemcpy(routing_table, device_routing_table_ptr, total_elem_num*sizeof(RoutingEntry), hipMemcpyDeviceToHost) );
            printBestPath(from, to, routing_table);
        }


        nodesTick<<<nodes_num, thread_num>>>(device_incomming_buffer_ptr, 
                                             device_outgoing_buffer_ptr, 
                                             device_routing_table_ptr,
                                             from, to, ticks);
        CudaCheckError();
        // hipDeviceSynchronize();
        
        std::swap(device_incomming_buffer_ptr, 
                  device_outgoing_buffer_ptr);
    }

    deinitializePacketBuffers<<<nodes_num, thread_num>>>(device_incomming_buffer_ptr, 
                                                         device_outgoing_buffer_ptr);
    CudaCheckError();


    // hipMemcpy(device_incomming_buffer_ptr, incomming_buffer, buffers_elem_num*sizeof(PacketBuffer), hipMemcpyDeviceToHost);
    // hipMemcpy(device_outgoing_buffer_ptr, outgoing_buffer, buffers_elem_num*sizeof(PacketBuffer), hipMemcpyDeviceToHost);
    CudaSafeCall( hipMemcpy(routing_table, device_routing_table_ptr, total_elem_num*sizeof(RoutingEntry), hipMemcpyDeviceToHost) );

    //check the results
    printBestPath(from, to, routing_table);
    // printRoutingTable(routing_table);

    CudaSafeCall( hipFree(device_incomming_buffer_ptr) );
    CudaSafeCall( hipFree(device_outgoing_buffer_ptr) );
    CudaSafeCall( hipFree(device_routing_table_ptr) );
}